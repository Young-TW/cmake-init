#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void hello_kernel() {
    printf("Hello from CUDA kernel!\n");
}

int main() {
    // Launch the kernel with 10 threads
    hello_kernel<<<1, 1>>>();

    // Wait for GPU to finish before accessing results
    hipDeviceSynchronize();

    return 0;
}