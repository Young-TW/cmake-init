#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void helloFromGPU() {
    printf("Hello from GPU thread %d\n", threadIdx.x);
}

int main() {
    // Launch the kernel with 10 threads
    helloFromGPU<<<1, 10>>>();

    // Wait for GPU to finish before accessing results
    hipDeviceSynchronize();

    return 0;
}